#include "hip/hip_runtime.h"
using namespace std;

#include <stdio.h>
#include <vector>
#include <iostream>
#include <fstream>
#include "cudaConvUtilities.h.cu"
using namespace std;

unsigned int cpuTimerVariable;
hipEvent_t eventTimerStart;
hipEvent_t eventTimerStop;

// Assume target memory has already been allocated, nPixels is odd
void createGaussian1D(float* targPtr, 
                      int    nPixels, 
                      float  sigma, 
                      float  ctr)
{
   if(nPixels%2 != 1)
   {
      cout << "***Warning: createGaussian(...) only defined for odd pixel"  << endl;
      cout << "            dimensions.  Undefined behavior for even sizes." << endl;
   }

   float pxCtr = (float)(nPixels/2 + ctr);   
   float sigmaSq = sigma*sigma;
   float denom = sqrt(2*M_PI*sigmaSq);
   float dist;
   for(int i=0; i<nPixels; i++)
   {
      dist = (float)i - pxCtr;
      targPtr[i] = exp(-0.5 * dist * dist / sigmaSq) / denom;
   }
}

// Assume target memory has already been allocate, nPixels is odd
// Use Row-Col (D00_UL_ES)
void createGaussian2D(float* targPtr, 
                      int    nPixelsRow,
                      int    nPixelsCol,
                      float  sigmaRow,
                      float  sigmaCol,
                      float  ctrRow,
                      float  ctrCol)
{
   if(nPixelsRow%2 != 1 || nPixelsCol != 1)
   {
      cout << "***Warning: createGaussian(...) only defined for odd pixel"  << endl;
      cout << "            dimensions.  Undefined behavior for even sizes." << endl;
   }

   float pxCtrRow = (float)(nPixelsRow/2 + ctrRow);   
   float pxCtrCol = (float)(nPixelsCol/2 + ctrCol);   
   float distRow, distCol, distRowSqNorm, distColSqNorm;
   float denom = 2*M_PI*sigmaRow*sigmaCol;
   for(int r=0; r<nPixelsRow; r++)
   {
      distRow = (float)r - pxCtrRow;
      distRowSqNorm = distRow*distRow / (sigmaRow*sigmaRow);
      for(int c=0; c<nPixelsCol; c++)
      {
         distCol = (float)c - pxCtrCol;
         distColSqNorm = distCol*distCol / (sigmaCol*sigmaCol);
         
         targPtr[r*nPixelsCol+c] = exp(-0.5*(distRowSqNorm + distColSqNorm)) / denom;
      }
   }
}


// Assume diameter^2 target memory has already been allocated
// This filter is used for edge detection.  Convolve with the
// kernel created by this function, and then look for the 
// zero-crossings
// As always, we expect an odd diameter
// For LoG kernels, we always assume square and symmetric,
// which is why there are no options for different dimensions
void createLaplacianOfGaussianKernel(float* targPtr,
                                     int    diameter)
{
   float pxCtr = (float)(diameter-1) / 2.0f;
   float dc, dr, dcSq, drSq;
   float sigma = diameter/10.0f;
   float sigmaSq = sigma*sigma;
   for(int r=0; r<diameter; r++)
   {
      dr = (float)r - pxCtr;
      drSq = dr*dr;
      for(int c=0; c<diameter; c++)
      {
         dc = (float)c - pxCtr;
         dcSq = dc*dc;
   
         float firstTerm  = (drSq + dcSq - 2*sigmaSq) / (sigmaSq * sigmaSq);
         float secondTerm = exp(-0.5 * (drSq + dcSq) / sigmaSq);
         targPtr[r*diameter+c] = firstTerm * secondTerm;
      }
   }
}

// Assume diameter^2 target memory has already been allocated
int createBinaryCircle(int* targPtr,
                       int  diameter)
{
   float pxCtr = (float)(diameter-1) / 2.0f;
   float rad;
   int seNonZero = 0;
   for(int r=0; r<diameter; r++)
   {
      for(int c=0; c<diameter; c++)
      {
         rad = sqrt((r-pxCtr)*(r-pxCtr) + (c-pxCtr)*(c-pxCtr));
         if(rad <= pxCtr+0.5)
         {
            targPtr[r*diameter+c] = 1;
            seNonZero++;
         }
         else
         {
            targPtr[r*diameter+c] = 0;
         }
      }
   }
   return seNonZero;
}

// Assume diameter^2 target memory has already been allocated
cudaImageHost<int> createBinaryCircle(int diameter)
{
   cudaImageHost<int> out(diameter, diameter);
   float pxCtr = (float)(diameter-1) / 2.0f;
   float rad;
   for(int r=0; r<diameter; r++)
   {
      for(int c=0; c<diameter; c++)
      {
         rad = sqrt((r-pxCtr)*(r-pxCtr) + (c-pxCtr)*(c-pxCtr));
         if(rad <= pxCtr+0.5)
            out(r,c) = 1.0f;
         else
            out(r,c) = 0.0f;
      }
   }
   return out;
}

////////////////////////////////////////////////////////////////////////////////
// Simple Timing Calls
void cpuStartTimer(void)
{
   // GPU Timer Functions
   cpuTimerVariable = 0;
   cutCreateTimer( &cpuTimerVariable );
   cutStartTimer(   cpuTimerVariable );
}

////////////////////////////////////////////////////////////////////////////////
// Stopping also resets the timer
// returns milliseconds
float cpuStopTimer(void)
{
   cutStopTimer( cpuTimerVariable );
   float cpuTime = cutGetTimerValue(cpuTimerVariable);
   cutDeleteTimer( cpuTimerVariable );
   return cpuTime;
}

////////////////////////////////////////////////////////////////////////////////
// Timing Calls for GPU -- this only counts GPU clock cycles, which will be 
// more precise for measuring GFLOPS and xfer rates, but shorter than wall time
void gpuStartTimer(void)
{
   hipEventCreate(&eventTimerStart);
   hipEventCreate(&eventTimerStop);
   hipEventRecord(eventTimerStart);
}

////////////////////////////////////////////////////////////////////////////////
// Stopping also resets the timer
float gpuStopTimer(void)
{
   hipEventRecord(eventTimerStop);
   hipEventSynchronize(eventTimerStop);
   float gpuTime;
   hipEventElapsedTime(&gpuTime, eventTimerStart, eventTimerStop);
   return gpuTime;
}

////////////////////////////////////////////////////////////////////////////////
// Read/Write images from/to files
void ReadFile(string fn, int* targPtr, int nRows, int nCols)
{
   ifstream in(fn.c_str(), ios::in);
   // We work with Row-Col format, but files written in Col-Row, so switch loop
   for(int r=0; r<nRows; r++)
      for(int c=0; c<nCols; c++)
         in >> targPtr[r*nRows+c];
   in.close();
}

////////////////////////////////////////////////////////////////////////////////
// Writing file in space-separated format
void WriteFile(string fn, int* srcPtr, int nRows, int nCols)
{
   ofstream out(fn.c_str(), ios::out);
   // We work with Row-Col format, but files written in Col-Row, so switch loop
   for(int r=0; r<nRows; r++)
   {
      for(int c=0; c<nCols; c++)
      {
         out << srcPtr[r*nCols+c] << " ";
      }
      out << endl;
   }
   out.close();
}

////////////////////////////////////////////////////////////////////////////////
// Writing image to stdout
void PrintArray(int* srcPtr, int nRows, int nCols)
{
   // We work with Row-Col format, but files written in Col-Row, so switch loop
   for(int r=0; r<nRows; r++)
   {
      cout << "\t";
      for(int c=0; c<nCols; c++)
      {
         cout << srcPtr[r*nCols+c] << " ";
      }
      cout << endl;
   }
}




////////////////////////////////////////////////////////////////////////////////
// Copy a 3D texture from a host (float*) array to a device hipArray
// The extent should be specified with all dimensions in units of *elements*
void prepareCudaTexture(float* h_src, 
                        hipArray *d_dst,
                        hipExtent const texExtent)
{
   hipMemcpy3DParms copyParams = {0};
   hipPitchedPtr cppImgPsf = make_hipPitchedPtr( (void*)h_src, 
                                                   texExtent.width*FLOAT_SZ,
                                                   texExtent.width,  
                                                   texExtent.height);
   copyParams.srcPtr   = cppImgPsf;
   copyParams.dstArray = d_dst;
   copyParams.extent   = texExtent;
   copyParams.kind     = hipMemcpyHostToDevice;
   cutilSafeCall( hipMemcpy3D(&copyParams) );
}
////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////
// BASIC UNARY & BINARY *MASK* OPERATORS
// 
// Could create LUTs, but I'm not sure the extra implementation complexity
// actually provides much benefit.  These ops already run on the order of
// microseconds.
//
// NOTE:  These operators are for images with {0,1}, only the MORPHOLOGICAL
//        operators will operate with {-1,0,1}
//
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
__global__ void  Mask_Union_Kernel( int* A, int* B, int* devOut)
{  
   const int idx = blockDim.x*blockIdx.x + threadIdx.x;

   if( A[idx] + B[idx] > 0)
      devOut[idx] = 1;
   else
      devOut[idx] = 0;
}

////////////////////////////////////////////////////////////////////////////////
__global__ void  Mask_Intersect_Kernel( int* A, int* B, int* devOut)
{  
   const int idx = blockDim.x*blockIdx.x + threadIdx.x;
   devOut[idx] = A[idx] * B[idx];
}

////////////////////////////////////////////////////////////////////////////////
// (A - B):   A is set to 0 if B is 1, otherwise A is left alone
__global__ void  Mask_Subtract_Kernel( int* A, int* B, int* devOut)
{  
   const int idx = blockDim.x*blockIdx.x + threadIdx.x;
   if( B[idx] == 0)
      devOut[idx] = A[idx];
   else 
      devOut[idx] = 0;
}

////////////////////////////////////////////////////////////////////////////////
// (A - B):   A is set to 0 if B is 1, otherwise A is left alone
__global__ void  Mask_Difference_Kernel( int* A, int* B, int* devOut)
{  
   const int idx = blockDim.x*blockIdx.x + threadIdx.x;
   
   if(A[idx] == B[idx])
      devOut[idx] = 0; 
   else
      devOut[idx] = 1; 

   // Should test if the extra algebra ops are worth removing the if-statement
   // Convert to {-1, +1}
   //int aval = A[idx]*2 - 1;
   //int bval = B[idx]*2 - 1;
   //devOut[idx] = (aval*bval+1)/2;
}

////////////////////////////////////////////////////////////////////////////////
__global__ void  Mask_Invert_Kernel( int* A, int* devOut)
{  
   const int idx = blockDim.x*blockIdx.x + threadIdx.x;
   devOut[idx] = 1 - A[idx];
}


////////////////////////////////////////////////////////////////////////////////
// TODO: This is a very dumb/slow equal operator, actually won't even work
//       Perhaps have the threads atomicAdd to a globalMem location if !=
//__global__ void  Mask_CountDiff_Kernel( int* A, int* B, int* globalMemCount)
//{  
   //const int idx = blockDim.x*blockIdx.x + threadIdx.x;
   //if(A[idx] != B[idx])
      //atomicAdd(numNotEqual, 1);
//}


////////////////////////////////////////////////////////////////////////////////
// TODO: Need to use reduction for this, but that can be kind of complicated
//       This operation destroys the input data, and the final result will be
//       stored in A[0]
__global__ void  Mask_Sum_Kernel( int* A, int valCount, int* scalarOut)
{  
   
   const int localIdx    = threadIdx.x;
   const int globalIdx   = blockDim.x*blockIdx.x + threadIdx.x;
   const int blockIdxOut = blockIdx.x / blockDim.x;

   while(valCount > 1)
   {
      int localCount = blockDim.x;
      while(localCount > 1)
      {
         localCount = localCount / 2;  
         if(localIdx < localCount)
            A[globalIdx] += A[globalIdx + localCount];
      }
   
      if(localIdx == 0)
         A[blockIdxOut] = A[globalIdx];

      valCount = valCount / blockDim.x;
   }

   if(globalIdx==0)
      scalarOut[0] = A[0];
}


////////////////////////////////////////////////////////////////////////////////
//
// This function takes an array of size N, and returns an array of size N/512
// that has the same sum as the original.  This method will need to be called
// recursively until the final size is one element that can be passed back to
// the host.
// 
// This kernel is not scalable.  I just assume that the block size will be 
// (256,1,1), so make sure you call it with that.  I did this to improve
// simplicity and speed slightly, at the expense of scalability

__global__ void  Image_SumReduceStep_Kernel( int* devBufIn,
                                             int* devBufOut,
                                             int  lastBlockSize)
{  
   // ONLY USE THIS FUNCTION WITH BLOCK SIZE = (256,1,1);
   // We write it for that to 
   __shared__ char sharedMem[4096];
   int* shmBuf1 = (int*)sharedMem;
   int* shmBuf2 = (int*)&sharedMem[512];

   int globalIdx = 512 * blockIdx.x + threadIdx.x;
   int localIdx  = threadIdx.x;

   shmBuf1[localIdx]     = 0;
   shmBuf1[localIdx+256] = 0;
   shmBuf2[localIdx]     = 0;
   shmBuf2[localIdx+256] = 0;

   if(blockIdx.x == gridDim.x-1)
   {
      if(localIdx+256 >= lastBlockSize) devBufIn[globalIdx+256] = 0;
      if(localIdx     >= lastBlockSize) devBufIn[globalIdx]     = 0;
   }

   // Now we reduce each block of 512 values (256 threads) to a single number

   shmBuf1[localIdx] = devBufIn[globalIdx] + devBufIn[globalIdx + 256]; __syncthreads();
   if(localIdx < 128) shmBuf2[localIdx] = shmBuf1[localIdx]+shmBuf1[localIdx+128]; __syncthreads();
   if(localIdx < 64)  shmBuf1[localIdx] = shmBuf2[localIdx]+shmBuf2[localIdx+64];  __syncthreads();
   if(localIdx < 32)  shmBuf2[localIdx] = shmBuf1[localIdx]+shmBuf1[localIdx+32];  __syncthreads();
   if(localIdx < 16)  shmBuf1[localIdx] = shmBuf2[localIdx]+shmBuf2[localIdx+16];  __syncthreads();
   if(localIdx < 8)   shmBuf2[localIdx] = shmBuf1[localIdx]+shmBuf1[localIdx+8];   __syncthreads();
   if(localIdx < 4)   shmBuf1[localIdx] = shmBuf2[localIdx]+shmBuf2[localIdx+4];   __syncthreads();
   if(localIdx < 2)   shmBuf2[localIdx] = shmBuf1[localIdx]+shmBuf1[localIdx+2];   __syncthreads();

   // 2 -> 1
   if(localIdx < 1)
      devBufOut[blockIdx.x] = shmBuf2[localIdx] + shmBuf2[localIdx + 1];
   __syncthreads(); 

}


// Yes, you really do need to pass in 2 full-sized, EXTRA, buffers
int Image_Sum(int* devImgToSum, int* devTemp1, int* devTemp2, int arraySize)
{
   // Yes, it seems silly to use two temp buffers to sum up an image, but
   // my goal was to make the reduction-kernel simple with the log(n) order of
   // gColth, but not necessarily space-efficient
   
   hipMemcpy(devTemp1, devImgToSum, arraySize*sizeof(int), hipMemcpyDeviceToDevice);
   int* buf1 = devTemp1;
   int* buf2 = devTemp2;
   int* bufTemp;

   // The reduction kernel geometry is hardcoded b/c I wanted the code to be 
   // simple, not necessarily scalable
   dim3 BLOCK(256,1,1);
   int nEltsLeft = arraySize;

   while(nEltsLeft > 1)
   {
      int nBlocks = (nEltsLeft-1)/512+1;
      int lastBlockSize = ((nEltsLeft - (nBlocks-1)*512 ) - 1) % 512 + 1;
      dim3 GRID(nBlocks, 1, 1);

      Image_SumReduceStep_Kernel<<<GRID,BLOCK>>>(buf1, buf2, lastBlockSize);

      bufTemp = buf1; 
      buf1    = buf2;
      buf2    = bufTemp;

      nEltsLeft = nBlocks;

      hipDeviceSynchronize();
   }

   // Seems silly to do a memcpy like this to get one number out of the device
   // but I'm not aware of any other way (there probably is)
   int output; 
   hipMemcpy(&output, buf1, sizeof(int), hipMemcpyDeviceToHost);
   return output;
}






